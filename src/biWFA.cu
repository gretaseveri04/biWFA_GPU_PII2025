#include "hip/hip_runtime.h"
extern "C" {
	#include "wavefront/wavefront_align.h"
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <fstream>
#include <vector>
#include "headers/commons.h"
#include "headers/biWFA.h"
#include <chrono>

#define CHECK(call)                                                                     \
{                                                                                     \
	const hipError_t err = call;                                                     \
	if (err != hipSuccess)                                                           \
	{                                                                                 \
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		exit(EXIT_FAILURE);                                                           \
	}                                                                                 \
}

#define CHECK_KERNELCALL()                                                            \
{                                                                                     \
	const hipError_t err = hipGetLastError();                                       \
	if (err != hipSuccess)                                                           \
	{                                                                                 \
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		exit(EXIT_FAILURE);                                                           \
	}                                                                                 \
}

__device__ void extend_max(bool *finish, const int score, int32_t *max_ak, wf_components_t *wf, const int max_score_scope, const int alignment_k, const int32_t alignment_offset, const int pattern_len) {
    if (wf->mwavefronts[score%num_wavefronts].offsets == NULL) {
        if (wf->alignment.num_null_steps > max_score_scope) {
            *finish = true;
        } else {
            *finish = false;
        }
    } else {
        // wavefront_extend_matches_packed_end2end_max()
        bool end_reached = false;
        int32_t max_antidiag_loc = 0;
        
        // Iterate over all wavefront offsets
        int k_start = wf->mwavefronts[score%num_wavefronts].lo;
        int k_end = wf->mwavefronts[score%num_wavefronts].hi;
        
        for (int k = k_start; k <= k_end; ++k) {
            int32_t offset = wf->mwavefronts[score%num_wavefronts].offsets[k];
            if (offset == OFFSET_NULL) continue;
            
            // wavefront_extend_matches_kernel_blockwise() or wavefront_extend_matches_kernel()
            int equal_chars = 0;
            for (int i = offset; i < pattern_len; i++) {
                if((i - k) >= 0 && (i - k) < pattern_len) {
                    if (wf->alignment.pattern[i - k] == wf->alignment.text[i]) {
                        equal_chars++;
                    } else break;
                }
            }
            offset += equal_chars;
            
            // Return extended offset
            wf->mwavefronts[score%num_wavefronts].offsets[k] = offset;
            
            // Calculate antidiagonal and update max if needed
            int32_t antidiag = (2 * offset) - k;
            if (max_antidiag_loc < antidiag) {
                max_antidiag_loc = antidiag;
            }
        }
        
        // Update the max antidiagonal location
        *max_ak = max_antidiag_loc;
        
        // wavefront_termination_end2end()
        if (wf->mwavefronts[score%num_wavefronts].lo > alignment_k || alignment_k > wf->mwavefronts[score%num_wavefronts].hi) {
            end_reached = false;
        } else {
            int32_t moffset = wf->mwavefronts[score%num_wavefronts].offsets[alignment_k];
            if (moffset < alignment_offset) {
                end_reached = false;
            } else {
                end_reached = true;
            }
        }
        
        *finish = end_reached;
    }
}

__device__ void extend(bool *finish, const int score, const wf_components_t *wf, const int max_score_scope, const int alignment_k, const int32_t alignment_offset, const int pattern_len) {
    wf_t *mwf = &wf->mwavefronts[score % num_wavefronts];
    
    if (mwf->offsets == NULL) {
        *finish = (wf->alignment.num_null_steps > max_score_scope);
        return;
    }

    int lo = mwf->lo;
    int hi = mwf->hi;
    int k = lo + threadIdx.x;

    int32_t offset = 0;
    if (k <= hi) {
        offset = mwf->offsets[k];

        for (int i = offset; i < pattern_len; ++i) {
            int pattern_pos = i - k;
            int text_pos = i;

            if (pattern_pos < 0 || pattern_pos >= pattern_len) break;
            if (wf->alignment.pattern[pattern_pos] != wf->alignment.text[text_pos]) break;

            ++offset;
        }

        mwf->offsets[k] = offset;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        bool end_reached = false;
        if (alignment_k >= lo && alignment_k <= hi) {
            int32_t moffset = mwf->offsets[alignment_k];
            end_reached = (moffset >= alignment_offset);
        }
        *finish = end_reached;
    }
}

__device__ void nextWF(int *score, wf_components_t *wf, const bool forward, const int max_score_scope, const int text_len, const int pattern_len, int32_t *matrix_wf_m_g, int32_t *matrix_wf_i_g, int32_t *matrix_wf_d_g) {
    // Compute next (s+1) wavefront
    ++(*score);

    int score_mod = *score%num_wavefronts;

    // wavefront_compute_affine()
    int mismatch = *score - penalty_mismatch;
    int gap_open = *score - penalty_gap_open - penalty_gap_ext;
    int gap_extend = *score - penalty_gap_ext;

    // wavefront_compute_get_mwavefront()
    if((*score / num_wavefronts) > 0) {
        // Resetting old wavefronts' values
        wf->mwavefronts[score_mod].lo = -1;
        wf->mwavefronts[score_mod].hi = 1;
        wf->iwavefronts[score_mod].lo = -1;
        wf->iwavefronts[score_mod].hi = 1;
        wf->dwavefronts[score_mod].lo = -1;
        wf->dwavefronts[score_mod].hi = 1;
    }
    wf->mwavefronts[score_mod].offsets = matrix_wf_m_g + (num_wavefronts * wf_length * blockIdx.x) + (score_mod * wf_length) + wf_length/2;
    wf->mwavefronts[score_mod].null = false;
    wf->iwavefronts[score_mod].offsets = matrix_wf_i_g + (num_wavefronts * wf_length * blockIdx.x) + (score_mod * wf_length) + wf_length/2;
    wf->iwavefronts[score_mod].null = false;
    wf->dwavefronts[score_mod].offsets = matrix_wf_d_g + (num_wavefronts * wf_length * blockIdx.x) + (score_mod * wf_length) + wf_length/2;
    wf->dwavefronts[score_mod].null = false;

    wf_t in_mwavefront_misms = (mismatch < 0 || wf->mwavefronts[mismatch%num_wavefronts].offsets == NULL || wf->mwavefronts[mismatch%num_wavefronts].null) ? wf->wavefront_null : wf->mwavefronts[mismatch%num_wavefronts];
    wf_t in_mwavefront_open = (gap_open < 0 || wf->mwavefronts[gap_open%num_wavefronts].offsets == NULL || wf->mwavefronts[gap_open%num_wavefronts].null) ? wf->wavefront_null : wf->mwavefronts[gap_open%num_wavefronts];
    wf_t in_iwavefront_ext = (gap_extend < 0 || wf->iwavefronts[gap_extend%num_wavefronts].offsets == NULL || wf->iwavefronts[gap_extend%num_wavefronts].null) ? wf->wavefront_null : wf->iwavefronts[gap_extend%num_wavefronts];
    wf_t in_dwavefront_ext = (gap_extend < 0 || wf->dwavefronts[gap_extend%num_wavefronts].offsets == NULL || wf->dwavefronts[gap_extend%num_wavefronts].null) ? wf->wavefront_null : wf->dwavefronts[gap_extend%num_wavefronts];

    if (in_mwavefront_misms.null && in_mwavefront_open.null && in_iwavefront_ext.null && in_dwavefront_ext.null) {
        // wavefront_compute_allocate_output_null()
        wf->alignment.num_null_steps++; // Increment null-steps
        // Nullify Wavefronts
        wf->mwavefronts[score_mod].null = true;
        wf->iwavefronts[score_mod].null = true;
        wf->dwavefronts[score_mod].null = true;
    } else {
        wf->alignment.num_null_steps = 0;
        int hi, lo;

        // wavefront_compute_limits_input()
        int min_lo = in_mwavefront_misms.lo;
        int max_hi = in_mwavefront_misms.hi;

        if (!in_mwavefront_open.null && min_lo > (in_mwavefront_open.lo - 1)) min_lo = in_mwavefront_open.lo - 1;
        if (!in_mwavefront_open.null && max_hi < (in_mwavefront_open.hi + 1)) max_hi = in_mwavefront_open.hi + 1;
        if (!in_iwavefront_ext.null && min_lo > (in_iwavefront_ext.lo + 1)) min_lo = in_iwavefront_ext.lo + 1;
        if (!in_iwavefront_ext.null && max_hi < (in_iwavefront_ext.hi + 1)) max_hi = in_iwavefront_ext.hi + 1;
        if (!in_dwavefront_ext.null && min_lo > (in_dwavefront_ext.lo - 1)) min_lo = in_dwavefront_ext.lo - 1;
        if (!in_dwavefront_ext.null && max_hi < (in_dwavefront_ext.hi - 1)) max_hi = in_dwavefront_ext.hi - 1;
        lo = min_lo;
        hi = max_hi;

        // wavefront_compute_allocate_output()
        int effective_lo = lo;
        int effective_hi = hi;

        // wavefront_compute_limits_output()
        int eff_lo = effective_lo - (max_score_scope + 1);
        int eff_hi = effective_hi + (max_score_scope + 1);
        effective_lo = MIN(eff_lo, wf->alignment.historic_min_lo);
        effective_hi = MAX(eff_hi, wf->alignment.historic_max_hi);
        wf->alignment.historic_min_lo = effective_lo;
        wf->alignment.historic_max_hi = effective_hi;

        // Allocate M-Wavefront
        wf->mwavefronts[score_mod].lo = lo;
        wf->mwavefronts[score_mod].hi = hi;
        // Allocate I1-Wavefront
        if (!in_mwavefront_open.null || !in_iwavefront_ext.null) {
            wf->iwavefronts[score_mod].lo = lo;
            wf->iwavefronts[score_mod].hi = hi;
        } else {
            wf->iwavefronts[score_mod].null = true;
        }
        // Allocate D1-Wavefront
        if (!in_mwavefront_open.null || !in_dwavefront_ext.null) {
            wf->dwavefronts[score_mod].lo = lo;
            wf->dwavefronts[score_mod].hi = hi;
        } else {
            wf->dwavefronts[score_mod].null = true;
        }

        // wavefront_compute_init_ends()
        // Init wavefront ends
        bool m_misms_null = in_mwavefront_misms.null;
        bool m_gap_null = in_mwavefront_open.null;
        bool i_ext_null = in_iwavefront_ext.null;
        bool d_ext_null = in_dwavefront_ext.null;

        if (!m_misms_null) {
            // wavefront_compute_init_ends_wf_higher()
            if (in_mwavefront_misms.wf_elements_init_max >= hi) {
            } else {
                // Initialize lower elements
                int max_init = MAX(in_mwavefront_misms.wf_elements_init_max, in_mwavefront_misms.hi);
                int k;
                int tidx = threadIdx.x; 
                int num_threads = blockDim.x; 

                for (int k = max_init + 1 + tidx; k <= hi; k += num_threads) {
                    in_mwavefront_misms.offsets[k] = OFFSET_NULL;
                }

                if (tidx == 0) {
                    in_mwavefront_misms.wf_elements_init_max = hi;
                }
            }
            // wavefront_compute_init_ends_wf_lower()
            if (in_mwavefront_misms.wf_elements_init_min <= lo) {
            } else {
                // Initialize lower elements
                int min_init = MIN(in_mwavefront_misms.wf_elements_init_min, in_mwavefront_misms.lo);
                int k;
                int tidx = threadIdx.x; 
                int num_threads = blockDim.x; 

                for (int k = lo + tidx; k < min_init; k += num_threads) {
                    in_mwavefront_misms.offsets[k] = OFFSET_NULL;
                }
                if (tidx == 0) {
                    in_mwavefront_misms.wf_elements_init_min = lo;
                }

            }
        }
        if (!m_gap_null) {
            // wavefront_compute_init_ends_wf_higher()
            if (in_mwavefront_open.wf_elements_init_max >= hi + 1) {
            } else {
                // Initialize lower elements
                int max_init = MAX(in_mwavefront_open.wf_elements_init_max, in_mwavefront_open.hi);
                int k;
                int tidx = threadIdx.x; 
                int num_threads = blockDim.x; 

                for (int k = max_init + 1 + tidx; k <= hi + 1; k += num_threads) {
                    in_mwavefront_open.offsets[k] = OFFSET_NULL;
                }
                if (tidx == 0) {
                    in_mwavefront_open.wf_elements_init_max = hi + 1;
                }

            }
            // wavefront_compute_init_ends_wf_lower()
            if (in_mwavefront_open.wf_elements_init_min <= lo - 1) {
            } else {
                // Initialize lower elements
                int min_init = MIN(in_mwavefront_open.wf_elements_init_min, in_mwavefront_open.lo);
                int k;
                int tidx = threadIdx.x;
                int num_threads = blockDim.x;

                for (int k = lo - 1 + tidx; k < min_init; k += num_threads) {
                    in_mwavefront_open.offsets[k] = OFFSET_NULL;
                }

                if (tidx == 0) {
                    in_mwavefront_open.wf_elements_init_min = lo - 1;
                }

            }
        }
        if (!i_ext_null) {
            // wavefront_compute_init_ends_wf_higher()
            if (in_iwavefront_ext.wf_elements_init_max >= hi) {
            } else {
                // Initialize lower elements
                int max_init = MAX(in_iwavefront_ext.wf_elements_init_max, in_iwavefront_ext.hi);
                int k;
                int tidx = threadIdx.x;
                int num_threads = blockDim.x;

                for (int k = max_init + 1 + tidx; k <= hi; k += num_threads) {
                    in_iwavefront_ext.offsets[k] = OFFSET_NULL;
                }

                if (tidx == 0) {
                    in_iwavefront_ext.wf_elements_init_max = hi;
                }

            }
            // wavefront_compute_init_ends_wf_lower()
            if (in_iwavefront_ext.wf_elements_init_min <= lo - 1) {
            } else {
                // Initialize lower elements
                int min_init = MIN(in_iwavefront_ext.wf_elements_init_min, in_iwavefront_ext.lo);
                int k;
                int tidx = threadIdx.x;
                int num_threads = blockDim.x;

                for (int k = lo - 1 + tidx; k < min_init; k += num_threads) {
                    in_iwavefront_ext.offsets[k] = OFFSET_NULL;
                }

                if (tidx == 0) {
                    in_iwavefront_ext.wf_elements_init_min = lo - 1;
                }

            }
        }
        if (!d_ext_null) {
            // wavefront_compute_init_ends_wf_higher()
            if (in_dwavefront_ext.wf_elements_init_max >= hi + 1) {
            } else {
                // Initialize lower elements
                int max_init = MAX(in_dwavefront_ext.wf_elements_init_max, in_dwavefront_ext.hi);
                int k;
                int tidx = threadIdx.x;
                int num_threads = blockDim.x;

                for (int k = max_init + 1 + tidx; k <= hi + 1; k += num_threads) {
                    in_dwavefront_ext.offsets[k] = OFFSET_NULL;
                }

                if (tidx == 0) {
                    in_dwavefront_ext.wf_elements_init_max = hi + 1;
                }

            }
            // wavefront_compute_init_ends_wf_lower()
            if (in_dwavefront_ext.wf_elements_init_min <= lo) {
            } else {
                // Initialize lower elements
                int min_init = MIN(in_dwavefront_ext.wf_elements_init_min, in_dwavefront_ext.lo);
                int k;
                int tidx = threadIdx.x;
                int num_threads = blockDim.x;
                
                for (int k = lo + tidx; k < min_init; k += num_threads) {
                    in_dwavefront_ext.offsets[k] = OFFSET_NULL;
                }
                
                if (tidx == 0) {
                    in_dwavefront_ext.wf_elements_init_min = lo;
                }

            }
        }

        //wavefront_compute_affine_idm()
        // Compute-Next kernel loop
        int tidx = threadIdx.x;
        for (int i = lo; i <= hi; i += blockDim.x) {
            int idx = tidx + i;
            if (idx <= hi) {
                // Update I1
                int32_t ins_o = in_mwavefront_open.offsets[idx - 1];
                int32_t ins_e = in_iwavefront_ext.offsets[idx - 1];
                int32_t ins = MAX(ins_o, ins_e) + 1;
                wf->iwavefronts[score_mod].offsets[idx] = ins;

                // Update D1
                int32_t del_o = in_mwavefront_open.offsets[idx + 1];
                int32_t del_e = in_dwavefront_ext.offsets[idx + 1];
                int32_t del = MAX(del_o, del_e);
                wf->dwavefronts[score_mod].offsets[idx] = del;

                // Update M
                int32_t misms = in_mwavefront_misms.offsets[idx] + 1;
                int32_t max = MAX(del, MAX(misms, ins));

                // Adjust offset out of boundaries
                uint32_t h = max;
                uint32_t v = max - idx;
                if (h > text_len) max = OFFSET_NULL;
                if (v > pattern_len) max = OFFSET_NULL;
                wf->mwavefronts[score_mod].offsets[idx] = max;
            }
        }

        // wavefront_compute_process_ends()
        if (wf->mwavefronts[score_mod].offsets) {
            // wavefront_compute_trim_ends()
            int k;
            int lo = wf->mwavefronts[score_mod].lo;
            for (k = wf->mwavefronts[score_mod].hi; k >= lo; --k) {
                // Fetch offset
                int32_t offset = wf->mwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->mwavefronts[score_mod].hi = k; // Set new hi
            wf->mwavefronts[score_mod].wf_elements_init_max = k;
            // Trim from lo
            int hi = wf->mwavefronts[score_mod].hi;
            for (k = wf->mwavefronts[score_mod].lo ; k <= hi; ++k) {
                // Fetch offset
                int32_t offset = wf->mwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->mwavefronts[score_mod].lo = k; // Set new lo
            wf->mwavefronts[score_mod].wf_elements_init_min = k;
            wf->mwavefronts[score_mod].null = (wf->mwavefronts[score_mod].lo > wf->mwavefronts[score_mod].hi);
        }
        if (wf->iwavefronts[score_mod].offsets) {
            // wavefront_compute_trim_ends()
            int k;
            int lo = wf->iwavefronts[score_mod].lo;
            for (k = wf->iwavefronts[score_mod].hi; k >= lo; --k) {
                // Fetch offset
                int32_t offset = wf->iwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->iwavefronts[score_mod].hi = k; // Set new hi
            wf->iwavefronts[score_mod].wf_elements_init_max = k;
            // Trim from lo
            int hi = wf->iwavefronts[score_mod].hi;
            for (k = wf->iwavefronts[score_mod].lo; k <= hi; ++k) {
                // Fetch offset
                int32_t offset = wf->iwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->iwavefronts[score_mod].lo = k; // Set new lo
            wf->iwavefronts[score_mod].wf_elements_init_min = k;
            wf->iwavefronts[score_mod].null = (wf->iwavefronts[score_mod].lo > wf->iwavefronts[score_mod].hi);
        }
        if (wf->dwavefronts[score_mod].offsets) {
            // wavefront_compute_trim_ends()
            int k;
            int lo = wf->dwavefronts[score_mod].lo;
            for (k = wf->dwavefronts[score_mod].hi; k >= lo ; --k) {
                // Fetch offset
                int32_t offset = wf->dwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->dwavefronts[score_mod].hi = k; // Set new hi
            wf->dwavefronts[score_mod].wf_elements_init_max = k;
            // Trim from lo
            int hi = wf->dwavefronts[score_mod].hi;
            for (k = wf->dwavefronts[score_mod].lo; k <= hi; ++k) {
                // Fetch offset
                int32_t offset = wf->dwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->dwavefronts[score_mod].lo = k; // Set new lo
            wf->dwavefronts[score_mod].wf_elements_init_min = k;
            wf->dwavefronts[score_mod].null = (wf->dwavefronts[score_mod].lo > wf->dwavefronts[score_mod].hi);
        }
    }
}

__device__ void breakpoint_indel2indel(const int score_0, const int score_1, const wf_t *dwf_0, const wf_t *dwf_1, int *breakpoint_score, const int text_len, const int pattern_len) {
    int lo_0 = dwf_0->lo;
    int hi_0 = dwf_0->hi;
    int lo_1 = text_len - pattern_len - dwf_1->hi;
    int hi_1 = text_len - pattern_len - dwf_1->lo;

    if (hi_1 < lo_0 || hi_0 < lo_1) return;

    int min_hi = min(hi_0, hi_1);
    int max_lo = max(lo_0, lo_1);

    __shared__ int local_min[NUM_THREADS];
    int tid = threadIdx.x;
    local_min[tid] = INT_MAX;

    for (int k_0 = max_lo + tid; k_0 <= min_hi; k_0 += NUM_THREADS) {
        int k_1 = text_len - pattern_len - k_0;
        int dh_0 = dwf_0->offsets[k_0];
        int dh_1 = dwf_1->offsets[k_1];

        if ((dh_0 + dh_1) >= text_len) {
            int candidate = score_0 + score_1 - penalty_gap_open;
            if (candidate < local_min[tid]) {
                local_min[tid] = candidate;
            }
        }
    }

    __syncthreads();

    if (tid == 0) {
        int min_val = INT_MAX;
        for (int i = 0; i < NUM_THREADS; i++) {
            if (local_min[i] < min_val) {
                min_val = local_min[i];
            }
        }

        if (min_val < *breakpoint_score) {
            *breakpoint_score = min_val;
        }
    }
}

__device__ void breakpoint_m2m(const int score_0, const int score_1, const wf_t *mwf_0, const wf_t *mwf_1, int *breakpoint_score, const int text_len, const int pattern_len) {
    // Check wavefronts overlapping
    int lo_0 = mwf_0->lo;
    int hi_0 = mwf_0->hi;
    int lo_1 = text_len - pattern_len - mwf_1->hi;
    int hi_1 = text_len - pattern_len - mwf_1->lo;

    if (hi_1 < lo_0 || hi_0 < lo_1) return;
    
    // Compute overlapping interval
    int min_hi = MIN(hi_0, hi_1);
    int max_lo = MAX(lo_0, lo_1);
    int k_0;
    for (k_0 = max_lo; k_0 <= min_hi; k_0++) {
        const int k_1 = text_len - pattern_len - k_0;
        // Fetch offsets
        const int mh_0 = mwf_0->offsets[k_0];
        const int mh_1 = mwf_1->offsets[k_1];
        // Check breakpoint m2m
        if (mh_0 + mh_1 >= text_len && score_0 + score_1 < *breakpoint_score) {
            *breakpoint_score = score_0 + score_1; 
            return;
        }
    }
}

__device__ void overlap(const int score_0, const wf_components_t *wf_0, const int score_1, const wf_components_t *wf_1, const int max_score_scope, int *breakpoint_score, const int text_len, const int pattern_len) {
    int score_mod_0 = score_0 % num_wavefronts;
    wf_t *mwf_0 = &wf_0->mwavefronts[score_mod_0];

    if (mwf_0 == NULL) return;
    wf_t *d1wf_0 = &wf_0->dwavefronts[score_mod_0];
    wf_t *i1wf_0 = &wf_0->iwavefronts[score_mod_0];

    int i;
    for (i = 0; i < max_score_scope; ++i) {
        const int score_i = score_1 - i;
        if (score_i < 0) break;
        int score_mod_i = score_i % num_wavefronts;

        if (score_0 + score_i - penalty_gap_open >= *breakpoint_score) continue;

        wf_t *d1wf_1 = &wf_1->dwavefronts[score_mod_i];
        if (d1wf_0 != NULL && d1wf_1 != NULL) {
            breakpoint_indel2indel(score_0, score_i, d1wf_0, d1wf_1, breakpoint_score, text_len, pattern_len);
        }

        wf_t *i1wf_1 = &wf_1->iwavefronts[score_mod_i];
        if (i1wf_0 != NULL && i1wf_1 != NULL) {
            breakpoint_indel2indel(score_0, score_i, i1wf_0, i1wf_1, breakpoint_score, text_len, pattern_len);
        }

        if (score_0 + score_i >= *breakpoint_score) continue;
        wf_t *mwf_1 = &wf_1->mwavefronts[score_mod_i];

        if (mwf_1 != NULL && mwf_0->offsets != NULL && mwf_1->offsets != NULL) {
            if (mwf_0->lo <= mwf_0->hi && mwf_1->lo <= mwf_1->hi) {
                breakpoint_m2m(score_0, score_i, mwf_0, mwf_1, breakpoint_score, text_len, pattern_len);
            }
        }
    }
}


__global__ void biWFA_kernel(char *pattern_concat_g, char *text_concat_g, char *pattern_r_concat_g, char *text_r_concat_g, int *pattern_lengths_g, int *text_lengths_g,
    int *pattern_offsets_g, int *text_offsets_g, int *breakpoint_score_g, wf_t *mwavefronts_f, wf_t *iwavefronts_f, wf_t *dwavefronts_f, wf_t *mwavefronts_r, wf_t *iwavefronts_r,
    wf_t *dwavefronts_r, const int lo_g, const int hi_g, int32_t *offsets_g, const int max_score_scope, int32_t *matrix_wf_m_f, int32_t *matrix_wf_i_f, int32_t *matrix_wf_d_f,
    int32_t *matrix_wf_m_r, int32_t *matrix_wf_i_r, int32_t *matrix_wf_d_r) 
{
    int alignment_id = blockIdx.x;
    int lo = lo_g;
    int hi = hi_g;

    int pattern_offset = pattern_offsets_g[alignment_id];
    int text_offset = text_offsets_g[alignment_id];
    int pattern_len = pattern_lengths_g[alignment_id];
    int text_len = text_lengths_g[alignment_id];

    char *pattern_f = pattern_concat_g + pattern_offset;
    char *text_f = text_concat_g + text_offset;
    char *pattern_r = pattern_r_concat_g + pattern_offset;
    char *text_r = text_r_concat_g + text_offset;

    int total_offsets_size = num_wavefronts * wf_length;
    for (int i = threadIdx.x; i < total_offsets_size; i += blockDim.x) {
        int base_idx = alignment_id * total_offsets_size + i;
        matrix_wf_m_f[base_idx] = OFFSET_NULL;
        matrix_wf_i_f[base_idx] = OFFSET_NULL;
        matrix_wf_d_f[base_idx] = OFFSET_NULL;
        matrix_wf_m_r[base_idx] = OFFSET_NULL;
        matrix_wf_i_r[base_idx] = OFFSET_NULL;
        matrix_wf_d_r[base_idx] = OFFSET_NULL;
    }
    __syncthreads();

    for (int i = threadIdx.x; i < num_wavefronts; i += blockDim.x) {
        int base_idx = alignment_id * num_wavefronts + i;
        wf_t *wfs[] = { &mwavefronts_f[base_idx], &iwavefronts_f[base_idx], &dwavefronts_f[base_idx],
                        &mwavefronts_r[base_idx], &iwavefronts_r[base_idx], &dwavefronts_r[base_idx] };
        for (int w = 0; w < 6; ++w) {
            wfs[w]->null = true;
            wfs[w]->lo = 0;
            wfs[w]->hi = 0;
            wfs[w]->offsets = NULL;
            wfs[w]->wf_elements_init_min = 0;
            wfs[w]->wf_elements_init_max = 0;
        }
    }
    __syncthreads();

    wf_components_t wf_f, wf_r;
    wf_alignment_t alignment_f, alignment_r;

    alignment_f.pattern = pattern_f;
    alignment_f.text = text_f;
    alignment_f.historic_max_hi = hi;
    alignment_f.historic_min_lo = lo;
    alignment_f.num_null_steps = 0;
    wf_f.alignment = alignment_f;

    alignment_r.pattern = pattern_r;
    alignment_r.text = text_r;
    alignment_r.historic_max_hi = hi;
    alignment_r.historic_min_lo = lo;
    alignment_r.num_null_steps = 0;
    wf_r.alignment = alignment_r;

    int wf_base_idx = num_wavefronts * alignment_id;
    wf_f.mwavefronts = mwavefronts_f + wf_base_idx;
    wf_f.iwavefronts = iwavefronts_f + wf_base_idx;
    wf_f.dwavefronts = dwavefronts_f + wf_base_idx;
    wf_r.mwavefronts = mwavefronts_r + wf_base_idx;
    wf_r.iwavefronts = iwavefronts_r + wf_base_idx;
    wf_r.dwavefronts = dwavefronts_r + wf_base_idx;

    for (int dir = 0; dir <= 1; dir++) {
        wf_components_t *wf = (dir == 0) ? &wf_f : &wf_r;
        int32_t *matrix_m = (dir == 0) ? matrix_wf_m_f : matrix_wf_m_r;
        int32_t *matrix_i = (dir == 0) ? matrix_wf_i_f : matrix_wf_i_r;
        int32_t *matrix_d = (dir == 0) ? matrix_wf_d_f : matrix_wf_d_r;

        wf->mwavefronts[0].offsets = matrix_m + alignment_id * total_offsets_size + 0 * wf_length + wf_length / 2;
        wf->iwavefronts[0].offsets = matrix_i + alignment_id * total_offsets_size + 0 * wf_length + wf_length / 2;
        wf->dwavefronts[0].offsets = matrix_d + alignment_id * total_offsets_size + 0 * wf_length + wf_length / 2;

        wf->mwavefronts[0].null = false;
        wf->mwavefronts[0].lo = -1;
        wf->mwavefronts[0].hi = 1;
        wf->mwavefronts[0].offsets[0] = 0;
        wf->mwavefronts[0].wf_elements_init_min = 0;
        wf->mwavefronts[0].wf_elements_init_max = 0;

        wf->iwavefronts[0].null = true;
        wf->dwavefronts[0].null = true;

        wf->wavefront_null.null = true;
        wf->wavefront_null.lo = 1;
        wf->wavefront_null.hi = -1;
        wf->wavefront_null.offsets = offsets_g + wf_length / 2;
    }

    int max_antidiag = text_len + pattern_len - 1;
    int score_f = 0, score_r = 0;
    int forward_max_ak = 0, reverse_max_ak = 0;
    int breakpoint_score = INT_MAX;
    bool finish = false;
    int alignment_k = text_len - pattern_len;

    extend_max(&finish, score_f, &forward_max_ak, &wf_f, max_score_scope, alignment_k, text_len, pattern_len);
    extend_max(&finish, score_r, &reverse_max_ak, &wf_r, max_score_scope, alignment_k, text_len, pattern_len);
    if (finish) return;

    int max_ak = 0;
    bool last_wf_forward = false;

    while (forward_max_ak + reverse_max_ak < max_antidiag) {
        nextWF(&score_f, &wf_f, true, max_score_scope, text_len, pattern_len, matrix_wf_m_f, matrix_wf_i_f, matrix_wf_d_f);
        extend_max(&finish, score_f, &max_ak, &wf_f, max_score_scope, alignment_k, text_len, pattern_len);
        forward_max_ak = max(forward_max_ak, max_ak);
        last_wf_forward = true;

        if (forward_max_ak + reverse_max_ak >= max_antidiag) break;

        nextWF(&score_r, &wf_r, false, max_score_scope, text_len, pattern_len, matrix_wf_m_r, matrix_wf_i_r, matrix_wf_d_r);
        extend_max(&finish, score_r, &max_ak, &wf_r, max_score_scope, alignment_k, text_len, pattern_len);
        reverse_max_ak = max(reverse_max_ak, max_ak);
        last_wf_forward = false;
    }

    int min_score_f, min_score_r;
    while (true) {
        if (last_wf_forward) {
            min_score_r = max(0, score_r - (max_score_scope - 1));
            if (score_f + min_score_r - penalty_gap_open >= breakpoint_score) break;
            overlap(score_f, &wf_f, score_r, &wf_r, max_score_scope, &breakpoint_score, text_len, pattern_len);
            nextWF(&score_r, &wf_r, true, max_score_scope, text_len, pattern_len, matrix_wf_m_r, matrix_wf_i_r, matrix_wf_d_r);
            extend(&finish, score_r, &wf_r, max_score_scope, alignment_k, text_len, pattern_len);
        }

        min_score_f = max(0, score_f - (max_score_scope - 1));
        if (min_score_f + score_r - penalty_gap_open >= breakpoint_score) break;
        overlap(score_r, &wf_r, score_f, &wf_f, max_score_scope, &breakpoint_score, text_len, pattern_len);
        nextWF(&score_f, &wf_f, false, max_score_scope, text_len, pattern_len, matrix_wf_m_f, matrix_wf_i_f, matrix_wf_d_f);
        extend(&finish, score_f, &wf_f, max_score_scope, alignment_k, text_len, pattern_len);

        if (score_r + score_f >= max_alignment_steps) break;
        last_wf_forward = true;
    }

    breakpoint_score = -breakpoint_score;
    breakpoint_score_g[alignment_id] = breakpoint_score;
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <input_file> <output_csv>\n", argv[0]);
        return 1;
    }

    FILE *fp = fopen(argv[1], "r");
    if (!fp) {
        perror("File open error");
        return 1;
    }

    FILE *csv_file = fopen(argv[2], "w");
    if (!csv_file) {
        perror("CSV file open error");
        fclose(fp);
        return 1;
    }

    int num_alignments, pattern_len, text_len;
    if (fscanf(fp, "%d %d %d\n", &num_alignments, &pattern_len, &text_len) != 3) {
        printf("Error reading header.\n");
        fclose(fp);
        fclose(csv_file);
        return 1;
    }

    int *gpu_scores = (int *)malloc(sizeof(int) * num_alignments);
    auto total_start = std::chrono::high_resolution_clock::now();

    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    size_t matrix_size_per_alignment = sizeof(int32_t) * num_wavefronts * wf_length;
    size_t estimated_required_mem_per_alignment = matrix_size_per_alignment * 12;
    size_t max_batch_size = 100;
    if (max_batch_size > 10000) max_batch_size = 10000;
    if (max_batch_size < 1) {
        fprintf(stderr, "Not enough memory for even a single alignment.\n");
        free(gpu_scores);
        fclose(fp);
        fclose(csv_file);
        return 1;
    }

    printf("GPU Memory - Free: %zu bytes\n", free_mem);

    int correct_alignments = 0; 
    char temp_buffer[1024];
    for (int offset = 0; offset < num_alignments; offset += max_batch_size) {
        int current_batch_size = (offset + max_batch_size <= num_alignments)
                                 ? max_batch_size
                                 : (num_alignments - offset);

        auto batch_start = std::chrono::high_resolution_clock::now(); 

        int *pattern_lengths = (int *)malloc(sizeof(int) * current_batch_size);
        int *text_lengths = (int *)malloc(sizeof(int) * current_batch_size);
        int *pattern_offsets = (int *)malloc(sizeof(int) * current_batch_size);
        int *text_offsets = (int *)malloc(sizeof(int) * current_batch_size);

        int total_pattern_len = pattern_len * current_batch_size;
        int total_text_len = text_len * current_batch_size;

        char *pattern_concat = (char *)malloc(total_pattern_len);
        char *text_concat = (char *)malloc(total_text_len);
        char *pattern_r_concat = (char *)malloc(total_pattern_len);
        char *text_r_concat = (char *)malloc(total_text_len);

        bool read_error = false;

        for (int i = 0; i < current_batch_size; ++i) {
            pattern_lengths[i] = pattern_len;
            text_lengths[i] = text_len;
            pattern_offsets[i] = i * pattern_len;
            text_offsets[i] = i * text_len;

            if (!fgets(temp_buffer, sizeof(temp_buffer), fp)) {
                printf("Error reading pattern for alignment %d\n", offset + i);
                read_error = true;
                break;
            }
            strncpy(pattern_concat + pattern_offsets[i], temp_buffer, pattern_len);

            if (!fgets(temp_buffer, sizeof(temp_buffer), fp)) {
                printf("Error reading text for alignment %d\n", offset + i);
                read_error = true;
                break;
            }
            strncpy(text_concat + text_offsets[i], temp_buffer, text_len);

            for (int j = 0; j < pattern_len; ++j)
                pattern_r_concat[pattern_offsets[i] + j] = pattern_concat[pattern_offsets[i] + pattern_len - 1 - j];
            for (int j = 0; j < text_len; ++j)
                text_r_concat[text_offsets[i] + j] = text_concat[text_offsets[i] + text_len - 1 - j];
        }

        if (read_error) {
            free(pattern_concat); 
            free(text_concat);
            free(pattern_r_concat); 
            free(text_r_concat);
            free(pattern_lengths); 
            free(text_lengths);
            free(pattern_offsets); 
            free(text_offsets);
            free(gpu_scores);
            fclose(fp);
            fclose(csv_file);
            return 1;
        }

        char *d_pattern_concat, *d_text_concat, *d_pattern_r_concat, *d_text_r_concat;
        int *d_pattern_lengths, *d_text_lengths, *d_pattern_offsets, *d_text_offsets, *d_breakpoint_score;
        int32_t *d_offsets;
        size_t matrix_size = matrix_size_per_alignment * current_batch_size;

        wf_t *d_mwavefronts_f, *d_iwavefronts_f, *d_dwavefronts_f;
        wf_t *d_mwavefronts_r, *d_iwavefronts_r, *d_dwavefronts_r;
        int32_t *d_matrix_wf_m_f, *d_matrix_wf_i_f, *d_matrix_wf_d_f;
        int32_t *d_matrix_wf_m_r, *d_matrix_wf_i_r, *d_matrix_wf_d_r;

        CHECK(hipMalloc(&d_pattern_concat, total_pattern_len));
        CHECK(hipMalloc(&d_text_concat, total_text_len));
        CHECK(hipMalloc(&d_pattern_r_concat, total_pattern_len));
        CHECK(hipMalloc(&d_text_r_concat, total_text_len));
        CHECK(hipMalloc(&d_pattern_lengths, sizeof(int) * current_batch_size));
        CHECK(hipMalloc(&d_text_lengths, sizeof(int) * current_batch_size));
        CHECK(hipMalloc(&d_pattern_offsets, sizeof(int) * current_batch_size));
        CHECK(hipMalloc(&d_text_offsets, sizeof(int) * current_batch_size));
        CHECK(hipMalloc(&d_breakpoint_score, sizeof(int) * current_batch_size));
        CHECK(hipMalloc(&d_offsets, sizeof(int32_t) * wf_length));

        CHECK(hipMalloc(&d_mwavefronts_f, matrix_size));
        CHECK(hipMalloc(&d_iwavefronts_f, matrix_size));
        CHECK(hipMalloc(&d_dwavefronts_f, matrix_size));
        CHECK(hipMalloc(&d_mwavefronts_r, matrix_size));
        CHECK(hipMalloc(&d_iwavefronts_r, matrix_size));
        CHECK(hipMalloc(&d_dwavefronts_r, matrix_size));
        CHECK(hipMalloc(&d_matrix_wf_m_f, matrix_size));
        CHECK(hipMalloc(&d_matrix_wf_i_f, matrix_size));
        CHECK(hipMalloc(&d_matrix_wf_d_f, matrix_size));
        CHECK(hipMalloc(&d_matrix_wf_m_r, matrix_size));
        CHECK(hipMalloc(&d_matrix_wf_i_r, matrix_size));
        CHECK(hipMalloc(&d_matrix_wf_d_r, matrix_size));

        CHECK(hipMemcpy(d_pattern_concat, pattern_concat, total_pattern_len, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_text_concat, text_concat, total_text_len, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_pattern_r_concat, pattern_r_concat, total_pattern_len, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_text_r_concat, text_r_concat, total_text_len, hipMemcpyHostToDevice)); // Corretto qui
        CHECK(hipMemcpy(d_pattern_lengths, pattern_lengths, sizeof(int) * current_batch_size, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_text_lengths, text_lengths, sizeof(int) * current_batch_size, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_pattern_offsets, pattern_offsets, sizeof(int) * current_batch_size, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_text_offsets, text_offsets, sizeof(int) * current_batch_size, hipMemcpyHostToDevice));

        int32_t *temp_offsets = (int32_t *)malloc(sizeof(int32_t) * wf_length);
        for (int i = 0; i < wf_length; ++i) temp_offsets[i] = OFFSET_NULL;
        CHECK(hipMemcpy(d_offsets, temp_offsets, sizeof(int32_t) * wf_length, hipMemcpyHostToDevice));
        free(temp_offsets);

        int max_score_scope = max(penalty_gap_open + penalty_gap_ext, penalty_mismatch) + 1;
        int hi = max_score_scope + 1;
        int lo = -max_score_scope - 1;

        dim3 blocks(current_batch_size);
        dim3 threads(NUM_THREADS);
        biWFA_kernel<<<blocks, threads>>>(d_pattern_concat, d_text_concat, d_pattern_r_concat, d_text_r_concat,
                                          d_pattern_lengths, d_text_lengths, d_pattern_offsets, d_text_offsets,
                                          d_breakpoint_score,
                                          d_mwavefronts_f, d_iwavefronts_f, d_dwavefronts_f,
                                          d_mwavefronts_r, d_iwavefronts_r, d_dwavefronts_r,
                                          lo, hi, d_offsets, max_score_scope,
                                          d_matrix_wf_m_f, d_matrix_wf_i_f, d_matrix_wf_d_f,
                                          d_matrix_wf_m_r, d_matrix_wf_i_r, d_matrix_wf_d_r);

        CHECK(hipGetLastError());
        CHECK(hipDeviceSynchronize());

        int *breakpoint_score = (int *)malloc(sizeof(int) * current_batch_size);
        CHECK(hipMemcpy(breakpoint_score, d_breakpoint_score, sizeof(int) * current_batch_size, hipMemcpyDeviceToHost));

        auto batch_end = std::chrono::high_resolution_clock::now(); 
        std::chrono::duration<double> batch_duration = batch_end - batch_start;
        double alignments_per_second = current_batch_size / batch_duration.count();

        printf("\nResults for batch %d-%d:\n", offset, offset + current_batch_size - 1);
        printf("Batch time: %.6f seconds | Alignments per second: %.2f\n", batch_duration.count(), alignments_per_second);

        wavefront_aligner_attr_t attributes = wavefront_aligner_attr_default;
        attributes.distance_metric = gap_affine;
        attributes.affine_penalties.mismatch = penalty_mismatch;
        attributes.affine_penalties.gap_opening = penalty_gap_open;
        attributes.affine_penalties.gap_extension = penalty_gap_ext;
        wavefront_aligner_t *wf_aligner = wavefront_aligner_new(&attributes);

        for (int check_idx = 0; check_idx < current_batch_size; ++check_idx) {
            const char *pattern = pattern_concat + pattern_offsets[check_idx];
            const char *text = text_concat + text_offsets[check_idx];
            wavefront_align(wf_aligner, pattern, pattern_len, text, text_len);
            int cpu_score = wf_aligner->cigar->score;
            int gpu_score = breakpoint_score[check_idx];

            printf("Alignment %d:\n", offset + check_idx);
            printf("CPU Score: %d\n", cpu_score);
            printf("GPU Score: %d\n", gpu_score);
            printf("Match: %s\n\n", (cpu_score == gpu_score) ? "YES" : "NO");

            if (cpu_score == gpu_score) {
                correct_alignments++; 
            }
        }

        wavefront_aligner_delete(wf_aligner);

        free(breakpoint_score);
        hipFree(d_pattern_concat); 
        hipFree(d_text_concat);
        hipFree(d_pattern_r_concat); 
        hipFree(d_text_r_concat);
        hipFree(d_pattern_lengths); 
        hipFree(d_text_lengths);
        hipFree(d_pattern_offsets); 
        hipFree(d_text_offsets);
        hipFree(d_breakpoint_score); 
        hipFree(d_offsets);
        hipFree(d_mwavefronts_f); 
        hipFree(d_iwavefronts_f); 
        hipFree(d_dwavefronts_f);
        hipFree(d_mwavefronts_r); 
        hipFree(d_iwavefronts_r); 
        hipFree(d_dwavefronts_r);
        hipFree(d_matrix_wf_m_f);
        hipFree(d_matrix_wf_i_f); 
        hipFree(d_matrix_wf_d_f);
        hipFree(d_matrix_wf_m_r); 
        hipFree(d_matrix_wf_i_r); 
        hipFree(d_matrix_wf_d_r);
        free(pattern_concat); 
        free(text_concat);
        free(pattern_r_concat); 
        free(text_r_concat);
        free(pattern_lengths); 
        free(text_lengths);
        free(pattern_offsets); 
        free(text_offsets);
    }

    auto total_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> total_duration = total_end - total_start;

    double alignments_per_second = num_alignments / total_duration.count();

    printf("\nTotal correct alignments: %d\n", correct_alignments);
    printf("Total execution time: %.3f seconds\n", total_duration.count());
    printf("Alignments per second: %.3f\n", alignments_per_second);

    fprintf(csv_file, "Total correct alignments,%d\n", correct_alignments);
    fprintf(csv_file, "Total execution time (seconds),%.3f\n", total_duration.count());
    fprintf(csv_file, "Alignments per second,%.3f\n", alignments_per_second);

    free(gpu_scores);
    fclose(fp);
    fclose(csv_file);
    return 0;
}